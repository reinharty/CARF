#include "hip/hip_runtime.h"
//
// Created by Yorrick on 28.04.2018.
//
// Functions for CARF with uint128_t.
//

#include <random>
#include "hpc_helpers.hpp" //https://github.com/JGU-HPC/parallelprogrammingbook/blob/master/include/hpc_helpers.hpp
#include "uint128_t.cu"

/**
 * Best settings for Tesla:
 * (no multiGPU / no streams)
 * NUMINPUTLINES 198647808 for max memory usage on Tesla
 * NUMTHREADS 128 for first kernel (89ms)
 * NUMTHREADS 608 for second kernel (1772.65ms) // 608 1772.65ms // 512 1798.48ms //256 1842.08ms // 768 1900ms //
 *
 * Best settings for Volta:
 * NUMINPUTLINES (190447616)
 * NUMTHREADS CARF: (352, 480, ) 768
 * NUMTHREADS popCount: 1024 (32 = 125ms to 1024 = 121)
 */
#define NUMINPUTLINES 198647808
#define NUMTHREADS (1024)

#define NUMTHREADS1 (128)
#define NUMTHREADS2 (608)

#define REPEATS (19000)
#define NUMBLOCKS 10


///////////////////////////////////////////////////////////////////////////////
//CARF - KERNEL
///////////////////////////////////////////////////////////////////////////////

/**
 * Amends a single Hamming Mask, removing spurious 0s.
 */
__device__ uint128_t SHMS(uint128_t RH, uint128_t RL, uint128_t GH, uint128_t GL){
    return ((((RH ^ GH) | (RL ^ GL))<<1 & ((RH ^ GH) | (RL ^ GL))>>1) | (((RH ^ GH) | (RL ^ GL))<<1 & ((RH ^ GH) | (RL ^ GL))>>2) | (((RH ^ GH) | (RL ^ GL))<<2 & ((RH ^ GH) | (RL ^ GL))>>1) | ((RH ^ GH) | (RL ^ GL)));
}

/**
 * Computes the final bit-vector / Hamming Mask for the given pair using SHMS.
 * Stores final bit-vector in HM_OUT.
 */
__global__ void CARF(uint128_t * RH, uint128_t * RL, uint128_t * GH, uint128_t * GL, uint128_t * HM_OUT){

    const size_t thid = blockDim.x*blockIdx.x + threadIdx.x;

    if(thid < NUMINPUTLINES){
        HM_OUT[thid] = SHMS(RH[thid], RL[thid], GH[thid], GL[thid]) &
                       SHMS((RH[thid]>>1), (RL[thid]>>1), GH[thid], GL[thid]) &
                       SHMS((RH[thid]>>2), (RL[thid]>>2), GH[thid], GL[thid]) &
                       SHMS((RH[thid]<<1), (RL[thid]<<1), GH[thid], GL[thid]) &
                       SHMS((RH[thid]<<2), (RL[thid]<<2), GH[thid], GL[thid]);
        //printf("%u\n", HM_OUT[thid]);
    }
}

/**
 * Computes the final bit-vector / Hamming Mask for the given pair using SHMS.
 * Stores final bit-vector in HM_OUT.
 * Allows a single thread to process REPEATS many pairs.
 */
__global__ void CARF_loop(uint128_t * RH, uint128_t * RL, uint128_t * GH, uint128_t * GL, uint128_t * HM_OUT){

    const uint64_t thid = (blockDim.x*blockIdx.x + threadIdx.x)*REPEATS;

    if(thid < NUMINPUTLINES){

        for(uint16_t i = 0; i<REPEATS; i++){
            HM_OUT[(thid+i)] = SHMS(RH[(thid+i)], RL[(thid+i)], GH[(thid+i)], GL[(thid+i)]) &
                               SHMS((RH[(thid+i)]>>1), (RL[(thid+i)]>>1), GH[(thid+i)], GL[(thid+i)]) &
                               SHMS((RH[(thid+i)]>>2), (RL[(thid+i)]>>2), GH[(thid+i)], GL[(thid+i)]) &
                               SHMS((RH[(thid+i)]<<1), (RL[(thid+i)]<<1), GH[(thid+i)], GL[(thid+i)]) &
                               SHMS((RH[(thid+i)]<<2), (RL[(thid+i)]<<2), GH[(thid+i)], GL[(thid+i)]);
        }
    }
}






///////////////////////////////////////////////////////////////////////////////
// Conservative Population Count - KERNEL
///////////////////////////////////////////////////////////////////////////////

/**
 * Implementation of the conservative population count for uint128_t.
 * The combination of IF-ELSE-conditions and computation which provided best speed.
 * Use this!
 * Output is stored in errorCount.
 */
__global__ void ConservativePopCount(uint128_t * hm, uint128_t * errorCount){

    const auto thid = blockDim.x * blockIdx.x + threadIdx.x;

    if(thid<NUMINPUTLINES){

        uint8_t streak = 0;
        uint8_t totalErrors = 0;

        for(uint8_t i = 0; i < 128; i++){

            streak = streak + (((hm[thid]>>i) & 1) != 0);
            if (i > 0 && streak > 0 and (((hm[thid]>>i) & 1) == 0) and ((hm[thid]>>(i-1) & 1) != 0)) {
                totalErrors += 1 +((streak + 1) / 3);
                streak = 0;
            } else if(i==127 and streak > 0 and (((hm[thid]>>i) & 1) != 0) and ((hm[thid]>>(i-1) & 1) != 0)){
                totalErrors += 1 +((streak + 1) / 3);
            }
        }
        errorCount[thid] = totalErrors;
    }
}


/**
 * Allows a single thread to process REPEATS-many pairs.
 */
__global__ void naive_ConservativePopCount_loop(uint128_t * hm, uint128_t * errorCount){

    const uint64_t thid = (blockDim.x*blockIdx.x + threadIdx.x)*REPEATS;

    if(thid<NUMINPUTLINES) {

        for (uint16_t j = 0; j < REPEATS; j++) {

            uint8_t streak = 0;
            uint8_t totalErrors = 0;

            for (uint8_t i = 0; i < 128; i++) {

                streak = streak + (((hm[thid] >> i) & 1) != 0);
                if (i > 0 && streak > 0 and (((hm[thid] >> i) & 1) == 0) and ((hm[thid] >> (i - 1) & 1) != 0)) {
                    totalErrors += 1 + ((streak + 1) / 3);
                    streak = 0;
                } else if (i == 127 and streak > 0 and (((hm[thid] >> i) & 1) != 0) and
                           ((hm[thid] >> (i - 1) & 1) != 0)) {
                    totalErrors += 1 + ((streak + 1) / 3);
                }
            }
            errorCount[thid] = totalErrors;
        }
    }
}

///////////////////////////////////////////////////////////////////////////////
// END OF KERNELS
///////////////////////////////////////////////////////////////////////////////


///////////////////////////////////////////////////////////////////////////////
// Single threaded / host functions
///////////////////////////////////////////////////////////////////////////////


///////////////////////////////////////////////////////////////////////////////
// Host version of CARF
///////////////////////////////////////////////////////////////////////////////

/**
 * Serial SHMS
 */
uint128_t serialSHMS(uint128_t RH, uint128_t RL, uint128_t GH, uint128_t GL) {
    return ((((RH ^ GH) | (RL ^ GL)) << 1 & ((RH ^ GH) | (RL ^ GL)) >> 1) |
            (((RH ^ GH) | (RL ^ GL)) << 1 & ((RH ^ GH) | (RL ^ GL)) >> 2) |
            (((RH ^ GH) | (RL ^ GL)) << 2 & ((RH ^ GH) | (RL ^ GL)) >> 1) | ((RH ^ GH) | (RL ^ GL)));
}

/**
 * Serial CARF
 */
void serial_CARF(uint128_t * RH, uint128_t * RL, uint128_t * GH, uint128_t * GL, uint128_t * serial_HM_OUT){


    for(size_t i = 0; i < NUMINPUTLINES; i++){

        serial_HM_OUT[i] = serialSHMS(RH[i], RL[i], GH[i], GL[i]) &
                           serialSHMS((RH[i]>>1), (RL[i]>>1), GH[i], GL[i]) &
                           serialSHMS((RH[i]>>2), (RL[i]>>2), GH[i], GL[i]) &
                           serialSHMS((RH[i]<<1), (RL[i]<<1), GH[i], GL[i]) &
                           serialSHMS((RH[i]<<2), (RL[i]<<2), GH[i], GL[i]);

    }
}

/**
 * Serial naive conservative population count.
 */
void serial_conservative_popcount(uint128_t * final_hm, uint128_t * errorCount) {
    for (size_t index = 0; index < NUMINPUTLINES; index++) {

        std::bitset<64> hm1 = final_hm[index].LEFT;
        std::bitset<64> hm2 = final_hm[index].RIGHT;
        std::bitset<128> hm;

        for(int i = 127; i>63; i--){
            hm.set(i, hm1[i]);
        }

        for(int i = 63; i >= 0; i--){
            hm.set(i, hm2[i]);
        }

        size_t totalErrors = 0;
        size_t streak = 0;

        for (uint8_t i = 0; i < 128; i++) {
            if (hm[i] == 1) {
                streak++;
            }
            if (i > 0 and hm[i] == 0 and hm[i - 1] == 1) {
                totalErrors += 1 + ((streak + 1) / 3);
                streak = 0;
            } else if (i == 127 and streak > 0 and ((hm[i]) == 1) and ((hm[i - 1] == 1))) {
                totalErrors += 1 + ((streak + 1) / 3);
            }
        }
        errorCount[index] = totalErrors;
    }
}

///////////////////////////////////////////////////////////////////////////////
// I/O-functions
///////////////////////////////////////////////////////////////////////////////

/**
 * Generates NUMININPUTLINES many completely random pairs in the given arrays.
 */
void generate_random_input(uint128_t * rh, uint128_t * rl, uint128_t * gh, uint128_t * gl){

    std::mt19937_64 gen (std::random_device{}());
    std::uint64_t randomNumber = gen();

    for(size_t i = 0; i < NUMINPUTLINES; i++){
        uint128_t a(gen(), gen());
        uint128_t b(gen(), gen());
        uint128_t c(gen(), gen());
        uint128_t d(gen(), gen());
        rh[i] = a;
        rl[i] = b;
        gh[i] = c;
        gl[i] = d;
    }

//    cout << "\nPrinting inputs:" << endl;
//    for (int i = 0; i < 7; i++){
//        cout << "rh["<< i <<"] "; rh[i].printBits();
//        cout << "rl["<< i <<"] "; rl[i].printBits();
//        cout << "gh["<< i <<"] "; gh[i].printBits();
//        cout << "gl["<< i <<"] "; gl[i].printBits();
//    }
//    cout << endl;

}

size_t serial_countExceedingThreshold(uint128_t * errorCount, uint8_t threshold){
    size_t negatives = 0;

    for(int i = 0; i < NUMINPUTLINES; i++){
        //errorCount[i].print();cout<<endl;
        if(errorCount[i]>threshold){
            negatives++;
        }
    }

    cout << "Negatives: " << negatives << endl;
    return negatives;
}

///////////////////////////////////////////////////////////////////////////////
// Debug functions
///////////////////////////////////////////////////////////////////////////////

/**
 * Compares the final bit-vectors computed by carf and serial carf.
 * Prints differing vectors to console.
 */
void compare_results(uint128_t * serial_hm_out, uint128_t * parallel_hm_out){
    size_t errorcount = 0;
    for(size_t i = 0; i < NUMINPUTLINES; i++){

        if(serial_hm_out[i]!=parallel_hm_out[i]){
            errorcount++;
            cout << "Error in entry " << i << ": serial: "; serial_hm_out[i].print(); cout << " parallel: "; parallel_hm_out[i].print(); cout <<endl;
        }
    }
    cout << "Found errors: " << errorcount << endl;
}

/**
 * Prints device info.
 */
void printDeviceInfo(){
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
}


///////////////////////////////////////////////////////////////////////////////
// Sequences
///////////////////////////////////////////////////////////////////////////////


/**
 * Test-Code for uint128_t
 */
void kernel128_test() {

    cout << "Starting kernel128 test." << endl;

    TIMERSTART(total_kernel_test);
    uint128_t *rh = new uint128_t[NUMINPUTLINES];
    uint128_t *rl = new uint128_t[NUMINPUTLINES];
    uint128_t *gh = new uint128_t[NUMINPUTLINES];
    uint128_t *gl = new uint128_t[NUMINPUTLINES];
    uint128_t *serial_hm_out = new uint128_t[NUMINPUTLINES];
    uint128_t *parallel_hm_out = new uint128_t[NUMINPUTLINES];

    uint128_t *RH = nullptr, *RL = nullptr, *GH = nullptr, *GL = nullptr;

    TIMERSTART(generate_random_input);
    generate_random_input(rh, rl, gh, gl);
    TIMERSTOP(generate_random_input);

    rh[0] = rl[0] = gh[0] = gl[0] = 0;

    TIMERSTART(hipMalloc);
    hipMalloc(&RH, sizeof(uint128_t) * NUMINPUTLINES);    CUERR;
    hipMalloc(&RL, sizeof(uint128_t) * NUMINPUTLINES);    CUERR;
    hipMalloc(&GH, sizeof(uint128_t) * NUMINPUTLINES);    CUERR;
    hipMalloc(&GL, sizeof(uint128_t) * NUMINPUTLINES);    CUERR;

    TIMERSTOP(hipMalloc);

    TIMERSTART(hipMemcpy);
    hipMemcpy(RH, rh, NUMINPUTLINES * sizeof(uint128_t), hipMemcpyHostToDevice);  CUERR;
    hipMemcpy(RL, rl, NUMINPUTLINES * sizeof(uint128_t), hipMemcpyHostToDevice);  CUERR;
    hipMemcpy(GH, gh, NUMINPUTLINES * sizeof(uint128_t), hipMemcpyHostToDevice);  CUERR;
    hipMemcpy(GL, gl, NUMINPUTLINES * sizeof(uint128_t), hipMemcpyHostToDevice);  CUERR;
    TIMERSTOP(hipMemcpy);


    TIMERSTART(kernel_CARF128);
    CARF<<<SDIV(NUMINPUTLINES, NUMTHREADS1), NUMTHREADS1>>>(RH, RL, GH, GL, RL);  CUERR;
    //CARF_loop<<<NUMBLOCKS, NUMTHREADS>>>(RH, RL, GH, GL, HM_OUT); CUERR;
    TIMERSTOP(kernel_CARF128);

    TIMERSTART(cudaMemcpy2);
    hipMemcpy(parallel_hm_out, RL, NUMINPUTLINES * sizeof(uint128_t), hipMemcpyDeviceToHost); CUERR;
    TIMERSTOP(cudaMemcpy2);

    TIMERSTART(serial_algorithm);
    serial_CARF(rh, rl, gh, gl, serial_hm_out);
    TIMERSTOP(serial_algorithm);

    TIMERSTART(comparison);
    compare_results(serial_hm_out, parallel_hm_out);
    TIMERSTOP(comparison);
    serial_hm_out[(NUMINPUTLINES-1)].print();
    cout << "=" << endl;
    parallel_hm_out[(NUMINPUTLINES-1)].print();
    cout << endl;

    uint128_t * serial_errorCount = new uint128_t[NUMINPUTLINES];
    uint128_t * parallel_errorCount = new uint128_t[NUMINPUTLINES];

    TIMERSTART(cudaMemcpy3);
    hipMemcpy(RH, parallel_errorCount, NUMINPUTLINES*sizeof(uint128_t), hipMemcpyHostToDevice); CUERR;
    TIMERSTOP(cudaMemcpy3);

    TIMERSTART(parallel_popCount);
    ConservativePopCount<<<SDIV(NUMINPUTLINES, NUMTHREADS2), NUMTHREADS2>>>(RL, RH); CUERR;
    //ConservativePopCount_loop<<<NUMBLOCKS, NUMTHREADS>>>(RL, RH); CUERR;
    TIMERSTOP(parallel_popCount);

    TIMERSTART(cudaMemcpy4);
    hipMemcpy(parallel_errorCount, RH, NUMINPUTLINES * sizeof(uint128_t), hipMemcpyDeviceToHost); CUERR;
    TIMERSTOP(cudaMemcpy4);

    TIMERSTART(serial_conservativePopC);
    serial_conservative_popcount(serial_hm_out, serial_errorCount);
    TIMERSTOP(serial_conservativePopC);



    TIMERSTART(comparison_errorrate);
    size_t s = serial_countExceedingThreshold(serial_errorCount, 2);
    size_t p = serial_countExceedingThreshold(parallel_errorCount, 2);

    cout << s << "=" << p << endl;
    serial_errorCount[(NUMINPUTLINES-1)].print();
    cout << "=";
    parallel_errorCount[(NUMINPUTLINES-1)].print();
    cout << endl;

    TIMERSTOP(comparison_errorrate);

    hipFree(RH);
    hipFree(RL);
    hipFree(GH);
    hipFree(GL);
    delete rh, rl, gh, gl, parallel_hm_out, serial_hm_out;

    TIMERSTOP(total_kernel_test);
}


/**
 *
 */
void kernel128_benchmark(){

    cout << "starting kernel 128 benchmark" << endl;

    //set the ID of the CUDA device
    hipSetDevice(0);   CUERR;
    hipDeviceReset();  CUERR;
    printDeviceInfo();  CUERR;

    TIMERSTART(mallocHost);
    uint128_t *rh = new uint128_t[NUMINPUTLINES];
    uint128_t *rl = new uint128_t[NUMINPUTLINES];
    uint128_t *gh = new uint128_t[NUMINPUTLINES];
    uint128_t *gl = new uint128_t[NUMINPUTLINES];
    uint128_t *RH = nullptr, *RL = nullptr, *GH = nullptr, *GL = nullptr;
    uint128_t * parallel_errorCount = new uint128_t[NUMINPUTLINES];
    TIMERSTOP(mallocHost);

    TIMERSTART(generateInput);
    generate_random_input(rh, rl, gh, gl);
    TIMERSTOP(generateInput);

    TIMERSTART(hipMalloc);
    hipMalloc(&RH, sizeof(uint128_t) * NUMINPUTLINES);    CUERR;
    hipMalloc(&RL, sizeof(uint128_t) * NUMINPUTLINES);    CUERR;
    hipMalloc(&GH, sizeof(uint128_t) * NUMINPUTLINES);    CUERR;
    hipMalloc(&GL, sizeof(uint128_t) * NUMINPUTLINES);    CUERR;
    TIMERSTOP(hipMalloc);

//    for(int i=32; i <=1024; i+=32) {

//        cout << i << endl;
    TIMERSTART(cudaMemcpyToDevice);
    hipMemcpy(RH, rh, NUMINPUTLINES * sizeof(uint128_t), hipMemcpyHostToDevice);  CUERR;
    hipMemcpy(RL, rl, NUMINPUTLINES * sizeof(uint128_t), hipMemcpyHostToDevice);  CUERR;
    hipMemcpy(GH, gh, NUMINPUTLINES * sizeof(uint128_t), hipMemcpyHostToDevice);  CUERR;
    hipMemcpy(GL, gl, NUMINPUTLINES * sizeof(uint128_t), hipMemcpyHostToDevice);  CUERR;
    TIMERSTOP(cudaMemcpyToDevice);


    //TIMERSTART(kernels);
    TIMERSTART(CARF128);
    CARF<<<SDIV(NUMINPUTLINES, NUMTHREADS), NUMTHREADS>>>(RH, RL, GH, GL, RL);
    //CARF_loop<<<NUMBLOCKS, NUMTHREADS>>>(RH, RL, GH, GL, RL);
    CUERR;
    TIMERSTOP(CARF128);
    hipDeviceSynchronize();
    TIMERSTART(parallel_popCount);
    ConservativePopCount<<<SDIV(NUMINPUTLINES, NUMTHREADS), NUMTHREADS>>>(RL, RH);
    //ConservativePopCount_loop<<<NUMBLOCKS, NUMTHREADS>>>(RL, RH);
    CUERR;
    TIMERSTOP(parallel_popCount);
    //TIMERSTOP(kernels);
//    }

    TIMERSTART(cudaMemcpyToHost);
    hipMemcpy(parallel_errorCount, RH, NUMINPUTLINES * sizeof(uint128_t), hipMemcpyDeviceToHost); CUERR;
    TIMERSTOP(cudaMemcpyToHost);

    TIMERSTART(countErrors);
    serial_countExceedingThreshold(parallel_errorCount, 2);
    TIMERSTOP(countErrors);

    hipFree(RH);
    hipFree(RL);
    hipFree(GH);
    hipFree(GL);
    delete rh, rl, gh, gl, parallel_errorCount;
}


///////////////////////////////////////////////////////////////////////////////
// main
///////////////////////////////////////////////////////////////////////////////


int main(int argc, char * argv[]) {

    cout << "NUMINPUTLINES: " << NUMINPUTLINES << endl << "REPEATS: " << REPEATS << endl << "NUMTHREADS: " << NUMTHREADS
         << endl << "NUMBLOCKS: " << NUMBLOCKS << endl;

//    for(int i = 1; i<5; i++){
//        cout << "Run: " << i << endl;
    //kernel128_test();
    kernel128_benchmark();
//        kernel128_benchmark(1024, 1024);//volta
//    }


//    uint128_t a = 1242;
//    uint128_t b(2323,14);
//    uint128_t d(2323,14);
//    uint128_t c = 1242;
//
//    b.printBits();
//    b=b>>0;
//    cout << endl;
//    b.printBits();
//    cout << endl;
//
//    c.printBits();
//    c=c<<128;
//    c.printBits();
//    cout << endl;
}